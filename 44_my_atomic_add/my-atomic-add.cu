#include "hip/hip_runtime.h"
#include "../include/freshman.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

/**
 * This example illustrates implementation of custom atomic operations using
 * CUDA's built-in atomicCAS function to implement atomic signed 32-bit integer
 * addition.
 **/

__device__ int myAtomicAdd(int *address, int incr)
{
    // Create an initial guess for the value stored at *address.
    int guess = *address;
    int oldValue = atomicCAS(address, guess, guess + incr);

    // Loop while the guess is incorrect.
    // if first atomicCAS succeed,pass the while
    while (oldValue != guess)
    {
        guess = oldValue;
        oldValue = atomicCAS(address, guess, guess + incr);
    }
    return oldValue;
}

__device__ float myAtomicAdd2(float* address, float incr)
{
    unsigned int* typeAddress = (unsigned int*)address;
 
    float currentVal = *address;
 
    unsigned int expected = __float2uint_rn(currentVal);
 
    unsigned int desired = __float2uint_rn(currentVal + incr);
 
    int oldIntValue = atomicCAS(typeAddress, expected, desired);
 
    while(oldIntValue != expected)
    {
        expected = oldIntValue;
 
        desired = __float2uint_rn(__uint2float_rn(oldIntValue) + incr);
        oldIntValue = atomicCAS(typeAddress, expected, desired);
 
    }
 
    return __uint2float_rn(oldIntValue);
}

__global__ void kernel(int *sharedInteger)
{
    myAtomicAdd(sharedInteger, 1);
}

int main(int argc, char **argv)
{
    int h_sharedInteger;
    int *d_sharedInteger;
    CHECK(hipMalloc((void **)&d_sharedInteger, sizeof(int)));
    CHECK(hipMemset(d_sharedInteger, 0x00, sizeof(int)));

    kernel<<<4, 128>>>(d_sharedInteger);

    CHECK(hipMemcpy(&h_sharedInteger, d_sharedInteger, sizeof(int),
                     hipMemcpyDeviceToHost));
    printf("4 x 128 increments led to value of %d\n", h_sharedInteger);

    hipFree(d_sharedInteger);

    return 0;
}
