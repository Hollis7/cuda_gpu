#include<hip/hip_runtime.h>
#include<stdio.h>
#include"freshman.h"
int main(int argc,char **argv){
    hipFuncCache_t cacheConfig=hipFuncCachePreferShared;
    CHECK(hipDeviceSetCacheConfig(cacheConfig));
    printf("hipDeviceSetCacheConfig %s\n",hipGetErrorString(hipDeviceSetCacheConfig(cacheConfig)));
    return 0;

}